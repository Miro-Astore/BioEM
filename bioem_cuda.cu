#include "hip/hip_runtime.h"
/* ++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
   < BioEM software for Bayesian inference of Electron Microscopy images>
   Copyright (C) 2017 Pilar Cossio, David Rohr, Fabio Baruffa, Markus Rampp,
        Luka Stanisic, Volker Lindenstruth and Gerhard Hummer.
   Max Planck Institute of Biophysics, Frankfurt, Germany.
   Frankfurt Institute for Advanced Studies, Goethe University Frankfurt,
   Germany.
   Max Planck Computing and Data Facility, Garching, Germany.

   Released under the GNU Public License, v3.
   See license statement for terms of distribution.

   ++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/

#define BIOEM_GPUCODE

#if defined(_WIN32)
#include <windows.h>
#endif

#include <iostream>
using namespace std;

#include "bioem_cuda_internal.h"
//#include "hip/hip_runtime_api.h"

#include "bioem_algorithm.h"

#define checkCudaErrors(error)                                                 \
  {                                                                            \
    if ((error) != hipSuccess)                                                \
    {                                                                          \
      printf("CUDA Error %d / %s (%s: %d)\n", error,                           \
             hipGetErrorString(error), __FILE__, __LINE__);                   \
      exit(1);                                                                 \
    }                                                                          \
  }

#ifdef DEBUG_GPU
#define printCudaDebugStart()                                                  \
  float time;                                                                  \
  time = 0.;                                                                   \
  hipEvent_t start, stop;                                                     \
  checkCudaErrors(hipEventCreate(&start));                                    \
  checkCudaErrors(hipEventCreate(&stop));                                     \
  checkCudaErrors(hipEventRecord(start, 0));
#define printCudaDebug(msg)                                                    \
  checkCudaErrors(hipEventRecord(stop, 0));                                   \
  checkCudaErrors(hipEventSynchronize(stop));                                 \
  checkCudaErrors(hipEventElapsedTime(&time, start, stop));                   \
  printf("\t\t\tGPU: %s %1.6f sec\n", msg, time / 1000);                       \
  checkCudaErrors(hipEventRecord(start, 0));

#else
#define printCudaDebugStart()
#define printCudaDebug(msg)
#endif

static const char *cufftGetErrorStrung(hipfftResult error)
{
  switch (error)
  {
    case HIPFFT_SUCCESS:
      return "HIPFFT_SUCCESS";

    case HIPFFT_INVALID_PLAN:
      return "HIPFFT_INVALID_PLAN";

    case HIPFFT_ALLOC_FAILED:
      return "HIPFFT_ALLOC_FAILED";

    case HIPFFT_INVALID_TYPE:
      return "HIPFFT_INVALID_TYPE";

    case HIPFFT_INVALID_VALUE:
      return "HIPFFT_INVALID_VALUE";

    case HIPFFT_INTERNAL_ERROR:
      return "HIPFFT_INTERNAL_ERROR";

    case HIPFFT_EXEC_FAILED:
      return "HIPFFT_EXEC_FAILED";

    case HIPFFT_SETUP_FAILED:
      return "HIPFFT_SETUP_FAILED";

    case HIPFFT_INVALID_SIZE:
      return "HIPFFT_INVALID_SIZE";

    case HIPFFT_UNALIGNED_DATA:
      return "HIPFFT_UNALIGNED_DATA";
  }
  return "UNKNOWN";
}

/* Handing CUDA Driver errors */

#define cuErrorCheck(call)                                                     \
  do                                                                           \
  {                                                                            \
    hipError_t __error__;                                                        \
    if ((__error__ = (call)) != hipSuccess)                                  \
    {                                                                          \
      printf("CUDA Driver Error %d / %s (%s %d)\n", __error__,                 \
             cuGetError(__error__), __FILE__, __LINE__);                       \
      return __error__;                                                        \
    }                                                                          \
  } while (false)

static const char *cuGetError(hipError_t result)
{
  switch (result)
  {
    case hipSuccess:
      return "No errors";
    case hipErrorInvalidValue:
      return "Invalid value";
    case hipErrorOutOfMemory:
      return "Out of memory";
    case hipErrorNotInitialized:
      return "Driver not initialized";
    case hipErrorDeinitialized:
      return "Driver deinitialized";
    case hipErrorProfilerDisabled:
      return "Profiler disabled";
    case hipErrorProfilerNotInitialized:
      return "Profiler not initialized";
    case hipErrorProfilerAlreadyStarted:
      return "Profiler already started";
    case hipErrorProfilerAlreadyStopped:
      return "Profiler already stopped";
    case hipErrorNoDevice:
      return "No CUDA-capable device available";
    case hipErrorInvalidDevice:
      return "Invalid device";
    case hipErrorInvalidImage:
      return "Invalid kernel image";
    case hipErrorInvalidContext:
      return "Invalid context";
    case hipErrorContextAlreadyCurrent:
      return "Context already current";
    case hipErrorMapFailed:
      return "Map failed";
    case hipErrorUnmapFailed:
      return "Unmap failed";
    case hipErrorArrayIsMapped:
      return "Array is mapped";
    case hipErrorAlreadyMapped:
      return "Already mapped";
    case hipErrorNoBinaryForGpu:
      return "No binary for GPU";
    case hipErrorAlreadyAcquired:
      return "Already acquired";
    case hipErrorNotMapped:
      return "Not mapped";
    case hipErrorNotMappedAsArray:
      return "Not mapped as array";
    case hipErrorNotMappedAsPointer:
      return "Not mapped as pointer";
    case hipErrorECCNotCorrectable:
      return "Uncorrectable ECC error";
    case hipErrorUnsupportedLimit:
      return "Unsupported hipLimit_t";
    case hipErrorContextAlreadyInUse:
      return "Context already in use";
    case hipErrorInvalidSource:
      return "Invalid source";
    case hipErrorFileNotFound:
      return "File not found";
    case hipErrorSharedObjectSymbolNotFound:
      return "Shared object symbol not found";
    case hipErrorSharedObjectInitFailed:
      return "Shared object initialization failed";
    case hipErrorOperatingSystem:
      return "Operating System call failed";
    case hipErrorInvalidHandle:
      return "Invalid handle";
    case hipErrorNotFound:
      return "Not found";
    case hipErrorNotReady:
      return "CUDA not ready";
    case hipErrorLaunchFailure:
      return "Launch failed";
    case hipErrorLaunchOutOfResources:
      return "Launch exceeded resources";
    case hipErrorLaunchTimeOut:
      return "Launch exceeded timeout";
    case CUDA_ERROR_LAUNCH_INCOMPATIBLE_TEXTURING:
      return "Launch with incompatible texturing";
    case hipErrorPeerAccessAlreadyEnabled:
      return "Peer access already enabled";
    case hipErrorPeerAccessNotEnabled:
      return "Peer access not enabled";
    case hipErrorSetOnActiveProcess:
      return "Primary context active";
    case hipErrorContextIsDestroyed:
      return "Context is destroyed";
    case hipErrorAssert:
      return "Device assert failed";
    case CUDA_ERROR_TOO_MANY_PEERS:
      return "Too many peers";
    case hipErrorHostMemoryAlreadyRegistered:
      return "Host memory already registered";
    case hipErrorHostMemoryNotRegistered:
      return "Host memory not registered";
    case hipErrorUnknown:
      return "Unknown error";
    default:
      return "Unknown error code";
  }
}

bioem_cuda::bioem_cuda()
{
  deviceInitialized = 0;
  GPUAsync = getenv("GPUASYNC") == NULL ? 1 : atoi(getenv("GPUASYNC"));
  GPUWorkload =
      getenv("GPUWORKLOAD") == NULL ? 100 : atoi(getenv("GPUWORKLOAD"));
  if (GPUWorkload == -1)
    GPUWorkload = 100;
  GPUDualStream =
      getenv("GPUDUALSTREAM") == NULL ? 1 : atoi(getenv("GPUDUALSTREAM"));
}

bioem_cuda::~bioem_cuda() { deviceExit(); }

__global__ void multComplexMap(const mycomplex_t *convmap,
                               const mycomplex_t *refmap, mycuComplex_t *out,
                               const int MapSize, const int maxParallelConv,
                               const int NumberRefMaps, const int Offset)
{
  int myConv = myBlockIdxX / NumberRefMaps;
  int myRef = myBlockIdxX - myConv * NumberRefMaps + Offset;
  const mycuComplex_t *myin = (mycuComplex_t *) &refmap[myRef * MapSize];
  const mycuComplex_t *myconv = (mycuComplex_t *) &convmap[myConv * MapSize];
  mycuComplex_t *myout = &out[myBlockIdxX * MapSize];
  for (int i = myThreadIdxX; i < MapSize; i += myBlockDimX)
  {
    mycuComplex_t val;
    const mycuComplex_t conv = myconv[i];
    const mycuComplex_t in = myin[i];

    val.x = conv.x * in.x + conv.y * in.y;
    val.y = conv.y * in.x - conv.x * in.y;
    myout[i] = val;
  }
}

__global__ void
cuDoRefMapsFFT(const int iOrient, const int iConv, const myfloat_t *lCC,
               const myparam5_t *comp_params, bioem_Probability pProb,
               const bioem_param_device param, const bioem_RefMap RefMap,
               const int maxRef, const int Offset)
{
  if (myBlockIdxX * myBlockDimX + myThreadIdxX >= maxRef)
    return;
  const int iRefMap = myBlockIdxX * myBlockDimX + myThreadIdxX + Offset;
  const myfloat_t *mylCC = &lCC[(myBlockIdxX * myBlockDimX + myThreadIdxX) *
                                param.NumberPixels * param.NumberPixels];
  doRefMapFFT(iRefMap, iOrient, iConv, comp_params->amp, comp_params->pha,
              comp_params->env, comp_params->sumC, comp_params->sumsquareC,
              mylCC, pProb, param, RefMap);
}

__global__ void
doRefMap_GPU_Parallel(const int iRefMap, const int iOrient, const int iConv,
                      const int maxParallelConv, const myfloat_t *lCC,
                      const myparam5_t *comp_params, myblockGPU_t *comp_block,
                      bioem_Probability pProb, const bioem_param_device param,
                      const bioem_RefMap RefMap, const int maxRef,
                      const int dispC)
{
  int myGlobalId = myBlockIdxX * myBlockDimX + myThreadIdxX;
  if (myGlobalId >= maxParallelConv * param.NtotDisp)
    return;
  int myConv = myGlobalId / param.NtotDisp;
  myGlobalId -= myConv * param.NtotDisp;
  int myX = myGlobalId / param.NxDisp;
  myGlobalId -= myX * param.NxDisp;
  int myY = myGlobalId;
  myGlobalId = myBlockIdxX * myBlockDimX + myThreadIdxX;

  int cent_x = (myX * param.GridSpaceCenter + dispC) % param.NumberPixels;
  int cent_y = (myY * param.GridSpaceCenter + dispC) % param.NumberPixels;
  int address = (myConv * maxRef * param.NumberPixels * param.NumberPixels) +
                (cent_x * param.NumberPixels + cent_y);
  myfloat_t value = (myfloat_t) lCC[address] /
                    (myfloat_t)(param.NumberPixels * param.NumberPixels);

  __shared__ myprob_t bestLogpro[CUDA_THREAD_MAX];
  __shared__ int bestId[CUDA_THREAD_MAX];
  __shared__ myprob_t sumExp[CUDA_THREAD_MAX];
  __shared__ myprob_t sumAngles[CUDA_THREAD_MAX];

  int nTotalThreads =
      ((maxParallelConv * param.NtotDisp) < ((myBlockIdxX + 1) * myBlockDimX)) ?
          ((maxParallelConv * param.NtotDisp) - (myBlockIdxX * myBlockDimX)) :
          myBlockDimX;
  int halfPoint = (nTotalThreads + 1) >> 1; // divide by two

  bioem_Probability_map &pProbMap = pProb.getProbMap(iRefMap);

  bestLogpro[myThreadIdxX] =
      calc_logpro(param, comp_params[myConv].amp, comp_params[myConv].pha,
                  comp_params[myConv].env, comp_params[myConv].sumC,
                  comp_params[myConv].sumsquareC, value,
                  RefMap.sum_RefMap[iRefMap], RefMap.sumsquare_RefMap[iRefMap]);
#ifdef DEBUG_PROB
  printf("\t\t\tProb: iRefMap %d, iOrient %d, iConv %d, "
         "cent_x %d, cent_y %d, address %d, value %f, logpro %f\n",
         iRefMap, iOrient, iConv, cent_x, cent_y, address, value,
         bestLogpro[myThreadIdxX]);
#endif
  bestId[myThreadIdxX] = myGlobalId;
  sumExp[myThreadIdxX] = exp(bestLogpro[myThreadIdxX] - pProbMap.Constoadd);
  if (param.writeAngles)
  {
    bioem_Probability_angle &pProbAngle = pProb.getProbAngle(iRefMap, iOrient);
    sumAngles[myThreadIdxX] =
        exp(bestLogpro[myThreadIdxX] - pProbAngle.ConstAngle);
  }
  __syncthreads();

  // Total number of active threads
  while (nTotalThreads > 1)
  {
    if (myThreadIdxX < (nTotalThreads >> 1))
    {
      // Get the shared value stored by another thread
      myprob_t temp = bestLogpro[myThreadIdxX + halfPoint];
      if (temp > bestLogpro[myThreadIdxX])
      {
        bestLogpro[myThreadIdxX] = temp;
        bestId[myThreadIdxX] = bestId[myThreadIdxX + halfPoint];
      }
      sumExp[myThreadIdxX] += sumExp[myThreadIdxX + halfPoint];
      if (param.writeAngles)
      {
        sumAngles[myThreadIdxX] += sumAngles[myThreadIdxX + halfPoint];
      }
    }
    __syncthreads();
    nTotalThreads = halfPoint;            // divide by two.
    halfPoint = (nTotalThreads + 1) >> 1; // divide by two
    // only the first half of the threads will be active.
  }
  if (myThreadIdxX == 0)
  {
    comp_block[myBlockIdxX].logpro = bestLogpro[0];
    comp_block[myBlockIdxX].id = bestId[0];
    comp_block[myBlockIdxX].sumExp = sumExp[0];
    if (param.writeAngles)
    {
      comp_block[myBlockIdxX].sumAngles = sumAngles[0];
    }
#ifdef DEBUG_PROB
    printf("\t\t\tProb block: iRefMap %d, iOrient %d, iConv %d, "
           "bestlogpro %f, bestId %d, sumExp %f\n",
           iRefMap, iOrient, iConv, bestLogpro[0], bestId[0], sumExp[0]);
#endif
  }
}

__global__ void
doRefMap_GPU_Reduce(const int iRefMap, const int iOrient, const int iConv,
                    const int maxParallelConv, const myfloat_t *lCC,
                    const myparam5_t *comp_params,
                    const myblockGPU_t *comp_block, bioem_Probability pProb,
                    const bioem_param_device param, const bioem_RefMap RefMap,
                    const int maxRef, const int dispC)
{

  __shared__ myprob_t bestLogpro[CUDA_THREAD_MAX];
  __shared__ int bestId[CUDA_THREAD_MAX];
  __shared__ myprob_t sumExp[CUDA_THREAD_MAX];
  __shared__ myprob_t sumAngles[CUDA_THREAD_MAX];

  // if it is the last block
  int nTotalThreads = myBlockDimX;
  int halfPoint = (nTotalThreads + 1) >> 1; // divide by two

  bioem_Probability_map &pProbMap = pProb.getProbMap(iRefMap);

  bestLogpro[myThreadIdxX] = comp_block[myThreadIdxX].logpro;
  bestId[myThreadIdxX] = comp_block[myThreadIdxX].id;
  sumExp[myThreadIdxX] = comp_block[myThreadIdxX].sumExp;
  if (param.writeAngles)
  {
    sumAngles[myThreadIdxX] = comp_block[myThreadIdxX].sumAngles;
  }
  __syncthreads();
  while (nTotalThreads > 1)
  {
    if (myThreadIdxX < (nTotalThreads >> 1))
    {
      // Get the shared value stored by another thread
      myfloat_t temp = bestLogpro[myThreadIdxX + halfPoint];
      if (temp > bestLogpro[myThreadIdxX])
      {
        bestLogpro[myThreadIdxX] = temp;
        bestId[myThreadIdxX] = bestId[myThreadIdxX + halfPoint];
      }
      sumExp[myThreadIdxX] += sumExp[myThreadIdxX + halfPoint];
      if (param.writeAngles)
      {
        sumAngles[myThreadIdxX] += sumAngles[myThreadIdxX + halfPoint];
      }
    }
    __syncthreads();
    nTotalThreads = halfPoint;            // divide by two.
    halfPoint = (nTotalThreads + 1) >> 1; // divide by two
    // only the first half of the threads will be active.
  }

  if (myThreadIdxX == 0)
  {
    pProbMap.Total += sumExp[0];
    if (pProbMap.Constoadd < bestLogpro[0])
    {
      pProbMap.Total *= exp(-bestLogpro[0] + pProbMap.Constoadd);
      pProbMap.Constoadd = bestLogpro[0];

      // ********** Getting parameters that maximize the probability ***********
      int myGlobalId = bestId[0];
      int myConv = myGlobalId / param.NtotDisp;
      myGlobalId -= myConv * param.NtotDisp;
      int myX = myGlobalId / param.NxDisp;
      myGlobalId -= myX * param.NxDisp;
      int myY = myGlobalId;

      int cent_x = (myX * param.GridSpaceCenter + dispC) % param.NumberPixels;
      int cent_y = (myY * param.GridSpaceCenter + dispC) % param.NumberPixels;
      int address =
          (myConv * maxRef * param.NumberPixels * param.NumberPixels) +
          (cent_x * param.NumberPixels + cent_y);
      myfloat_t value = (myfloat_t) lCC[address] /
                        (myfloat_t)(param.NumberPixels * param.NumberPixels);

      pProbMap.max.max_prob_cent_x =
          -((myX * param.GridSpaceCenter + dispC) - param.NumberPixels);
      pProbMap.max.max_prob_cent_y =
          -((myY * param.GridSpaceCenter + dispC) - param.NumberPixels);
      pProbMap.max.max_prob_orient = iOrient;
      pProbMap.max.max_prob_conv = iConv + myConv;
      pProbMap.max.max_prob_norm =
          -(-comp_params[myConv].sumC * RefMap.sum_RefMap[iRefMap] +
            param.Ntotpi * value) /
          (comp_params[myConv].sumC * comp_params[myConv].sumC -
           comp_params[myConv].sumsquareC * param.Ntotpi);
      pProbMap.max.max_prob_mu =
          -(-comp_params[myConv].sumC * value +
            comp_params[myConv].sumsquareC * RefMap.sum_RefMap[iRefMap]) /
          (comp_params[myConv].sumC * comp_params[myConv].sumC -
           comp_params[myConv].sumsquareC * param.Ntotpi);

#ifdef DEBUG_PROB
      printf("\tProbabilities change: iRefMap %d, iOrient %d, iConv %d, "
             "Total %f, Const %f, bestlogpro %f, sumExp %f, bestId %d\n",
             iRefMap, iOrient, iConv + myConv, pProbMap.Total,
             pProbMap.Constoadd, bestLogpro[0], sumExp[0], bestId[0]);
      printf("\tParameters: myConv %d, myX %d, myY %d, cent_x %d, cent_y %d, "
             "probX %d, probY %d\n",
             myConv, myX, myY, cent_x, cent_y, pProbMap.max.max_prob_cent_x,
             pProbMap.max.max_prob_cent_y);
#endif
    }
#ifdef DEBUG_PROB
    printf("\t\tProbabilities after Reduce: iRefMap %d, iOrient %d, iConv "
           "%d, Total %f, Const %f, bestlogpro %f, sumExp %f, bestId %d\n",
           iRefMap, iOrient, iConv, pProbMap.Total, pProbMap.Constoadd,
           bestLogpro[0], sumExp[0], bestId[0]);
#endif

    if (param.writeAngles)
    {
      bioem_Probability_angle &pProbAngle =
          pProb.getProbAngle(iRefMap, iOrient);
      pProbAngle.forAngles += sumAngles[0];
      if (pProbAngle.ConstAngle < bestLogpro[0])
      {
        pProbAngle.forAngles *= exp(-bestLogpro[0] + pProbAngle.ConstAngle);
        pProbAngle.ConstAngle = bestLogpro[0];
      }
    }
  }
}

__global__ void
init_Constoadd(const int iRefMap, const int iOrient, const myfloat_t *lCC,
               const myparam5_t *comp_params, bioem_Probability pProb,
               const bioem_param_device param, const bioem_RefMap RefMap,
               const int initialized_const)
{
  myfloat_t value =
      (myfloat_t) lCC[0] / (myfloat_t)(param.NumberPixels * param.NumberPixels);

  myfloat_t logpro =
      calc_logpro(param, comp_params->amp, comp_params->pha, comp_params->env,
                  comp_params->sumC, comp_params->sumsquareC, value,
                  RefMap.sum_RefMap[iRefMap], RefMap.sumsquare_RefMap[iRefMap]);

  bioem_Probability_map &pProbMap = pProb.getProbMap(iRefMap);

  // Needed only once, in the first projection
  if (!initialized_const)
  {
    pProbMap.Constoadd = logpro;
  }
  // Needed for every projection
  if (param.writeAngles)
  {
    bioem_Probability_angle &pProbAngle = pProb.getProbAngle(iRefMap, iOrient);
    pProbAngle.ConstAngle = logpro;
  }

#ifdef DEBUG_GPU
  printf("\tInitialized pProbMap.Constoadd of refmap %d to %f\n", iRefMap,
         pProbMap.Constoadd);
#endif
}

template <class T> static inline T divup(T num, T divider)
{
  return ((num + divider - 1) / divider);
}

int bioem_cuda::compareRefMaps(int iPipeline, int iOrient, int iConv,
                               int maxParallelConv, mycomplex_t *conv_mapsFFT,
                               myparam5_t *comp_params, const int startMap)
{
  if (startMap)
  {
    cout << "Error startMap not implemented for GPU Code\n";
    exit(1);
  }
  printCudaDebugStart();
  if (GPUAsync)
  {
    checkCudaErrors(hipEventSynchronize(cudaEvent[iPipeline & 1]));
    printCudaDebug("time to synch projections");
  }

  int k = (iPipeline & 1) * param.nTotParallelConv;
  memcpy(&pConvMapFFT_Host[k * param.FFTMapSize],
         conv_mapsFFT[k * param.FFTMapSize],
         param.FFTMapSize * maxParallelConv * sizeof(mycomplex_t));
  printCudaDebug("time for memcpy");
  checkCudaErrors(
      hipMemcpyAsync(&pConvMapFFT[k * param.FFTMapSize],
                      &pConvMapFFT_Host[k * param.FFTMapSize],
                      param.FFTMapSize * maxParallelConv * sizeof(mycomplex_t),
                      hipMemcpyHostToDevice, cudaStream[GPUAsync ? 2 : 0]));
  // If one wants just a single tranfer, without memcpy:
  // checkCudaErrors(hipMemcpyAsync(&pConvMapFFT[k * param.FFTMapSize],
  // conv_mapsFFT[k * param.FFTMapSize], param.FFTMapSize * maxParallelConv *
  // sizeof(mycomplex_t), hipMemcpyHostToDevice, cudaStream[GPUAsync ? 2 :
  // 0]));
  checkCudaErrors(hipMemcpyAsync(&pTmp_comp_params[k], &comp_params[k],
                                  maxParallelConv * sizeof(myparam5_t),
                                  hipMemcpyHostToDevice,
                                  cudaStream[GPUAsync ? 2 : 0]));
  printCudaDebug("time for asyncmemcpy");
  if (GPUAsync)
  {
    checkCudaErrors(hipEventRecord(cudaEvent[2], cudaStream[2]));
    checkCudaErrors(hipStreamWaitEvent(cudaStream[0], cudaEvent[2], 0));
  }
  if (GPUDualStream)
  {
    checkCudaErrors(hipEventRecord(cudaFFTEvent[0], cudaStream[0]));
    checkCudaErrors(hipStreamWaitEvent(cudaStream[1], cudaFFTEvent[0], 0));
  }
  for (int offset = 0, stream = 0; offset < maxRef;
       offset += param.nTotParallelMaps, stream++)
  {
    if (!GPUDualStream)
      stream = 0;
    const int nRef = min(param.nTotParallelMaps, maxRef - offset);
    multComplexMap<<<maxParallelConv * nRef, CudaThreadCount, 0,
                     cudaStream[stream & 1]>>>(
        &pConvMapFFT[k * param.FFTMapSize], pRefMapsFFT, pFFTtmp2[stream & 1],
        param.FFTMapSize, maxParallelConv, nRef, offset);
    printCudaDebug("time for multComplexMap kernel");
    hipfftResult err = mycufftExecC2R(offset + param.nTotParallelMaps > maxRef ?
                                         plan[1][stream & 1] :
                                         plan[0][stream & 1],
                                     pFFTtmp2[stream & 1], pFFTtmp[stream & 1]);
    if (err != HIPFFT_SUCCESS)
    {
      cout << "Error running CUFFT " << cufftGetErrorStrung(err) << "\n";
      exit(1);
    }
    printCudaDebug("time for mycufftExecC2R kernel");
    if (BioEMAlgo == 1)
    {
      for (int conv = 0; conv < maxParallelConv; conv++)
      {
        cuDoRefMapsFFT<<<divup(nRef, CudaThreadCount), CudaThreadCount, 0,
                         cudaStream[stream & 1]>>>(
            iOrient, iConv + conv,
            pFFTtmp[stream & 1] +
                conv * nRef * param.param_device.NumberPixels *
                    param.param_device.NumberPixels,
            &pTmp_comp_params[k + conv], pProb_device, param.param_device,
            *gpumap, nRef, offset);
        printCudaDebug("time for cuDoRefMapsFFT kernel");
      }
    }
    else
    {
      for (int refmap = offset; refmap < nRef + offset; refmap++)
      {
        // First iteration needs to initialize Constoadd with the first valid
        // value to avoid overflow due to high sumExp values
        if ((initialized_const[refmap] == false) ||
            (param.param_device.writeAngles && iConv == 0))
        {
          init_Constoadd<<<1, 1, 0, cudaStream[stream & 1]>>>(
              refmap, iOrient,
              pFFTtmp[stream & 1] +
                  (refmap - offset) * param.param_device.NumberPixels *
                      param.param_device.NumberPixels,
              &pTmp_comp_params[k], pProb_device, param.param_device, *gpumap,
              (int) initialized_const[refmap]);
          initialized_const[refmap] = true;
          printCudaDebug("time for init_Constoadd kernel");
        }

        doRefMap_GPU_Parallel<<<divup(maxParallelConv *
                                          param.param_device.NtotDisp,
                                      CudaThreadCount),
                                CudaThreadCount, 0, cudaStream[stream & 1]>>>(
            refmap, iOrient, iConv, maxParallelConv,
            pFFTtmp[stream & 1] +
                (refmap - offset) * param.param_device.NumberPixels *
                    param.param_device.NumberPixels,
            &pTmp_comp_params[k], &pTmp_comp_blocks[refmap * Ncomp_blocks],
            pProb_device, param.param_device, *gpumap, nRef,
            param.param_device.NumberPixels -
                param.param_device.maxDisplaceCenter);
        printCudaDebug("time for doRefMaps_GPU_Parallel kernel");

        doRefMap_GPU_Reduce<<<1, divup(maxParallelConv *
                                           param.param_device.NtotDisp,
                                       CudaThreadCount),
                              0, cudaStream[stream & 1]>>>(
            refmap, iOrient, iConv, maxParallelConv,
            pFFTtmp[stream & 1] +
                (refmap - offset) * param.param_device.NumberPixels *
                    param.param_device.NumberPixels,
            &pTmp_comp_params[k], &pTmp_comp_blocks[refmap * Ncomp_blocks],
            pProb_device, param.param_device, *gpumap, nRef,
            param.param_device.NumberPixels -
                param.param_device.maxDisplaceCenter);
        printCudaDebug("time for doRefMaps_GPU_Reduce kernel");
      }
    }
  }
  checkCudaErrors(hipPeekAtLastError());

  if (GPUDualStream)
  {
    checkCudaErrors(hipEventRecord(cudaFFTEvent[1], cudaStream[1]));
    checkCudaErrors(hipStreamWaitEvent(cudaStream[0], cudaFFTEvent[1], 0));
  }

  if ((BioEMAlgo == 1) && (GPUWorkload < 100))
  {
    bioem::compareRefMaps(iPipeline, iOrient, iConv, maxParallelConv,
                          conv_mapsFFT, comp_params, maxRef);
    printCudaDebug("time to run OMP");
  }
  if (GPUAsync)
  {
    checkCudaErrors(hipEventRecord(cudaEvent[iPipeline & 1], cudaStream[0]));
  }
  else
  {
    checkCudaErrors(hipStreamSynchronize(cudaStream[0]));
    printCudaDebug("time to synch at the end");
  }
  return (0);
}

int bioem_cuda::selectCudaDevice()
{
  int count;
  int bestDevice = 0;
  hipDeviceProp_t deviceProp;

  /* Initializing CUDA driver API */
  cuErrorCheck(hipInit(0));

  /* Get number of available CUDA devices */
  checkCudaErrors(hipGetDeviceCount(&count));
  if (count == 0)
  {
    printf("No CUDA device detected\n");
    return (1);
  }

  /* Find the best GPU */
  long long int bestDeviceSpeed = -1, deviceSpeed = -1;
  for (int i = 0; i < count; i++)
  {
    hipGetDeviceProperties(&deviceProp, i);
    deviceSpeed = (long long int) deviceProp.multiProcessorCount *
                  (long long int) deviceProp.clockRate *
                  (long long int) deviceProp.warpSize;
    if (deviceSpeed > bestDeviceSpeed)
    {
      bestDevice = i;
      bestDeviceSpeed = deviceSpeed;
    }
  }

  /* Get user-specified GPU choice */
  if (getenv("GPUDEVICE"))
  {
    int device = atoi(getenv("GPUDEVICE"));
    if (device > count)
    {
      printf("Invalid CUDA device specified, max device number is %d\n", count);
      exit(1);
    }
#ifdef WITH_MPI
    if (device == -1)
    {
      device = mpi_rank % count;
    }
#endif
    if (device < 0)
    {
      printf("Negative CUDA device specified: %d, invalid!\n", device);
      exit(1);
    }
    bestDevice = device;
  }

  /* Set CUDA processes to appropriate devices */
  hipGetDeviceProperties(&deviceProp, bestDevice);
  if (deviceProp.computeMode == 0)
  {
    checkCudaErrors(hipSetDevice(bestDevice));
  }
  else
  {
    if (DebugOutput >= 1)
    {
      printf("CUDA device %d is not set in DEFAULT mode, make sure that CUDA "
             "processes are pinned as planned!\n",
             bestDevice);
      printf("Pinning process %d to CUDA device %d\n", mpi_rank, bestDevice);
    }
    checkCudaErrors(hipSetDevice(bestDevice));
    /* This synchronization is needed in order to detect bogus silent errors
     * from hipSetDevice call */
    checkCudaErrors(hipDeviceSynchronize());
  }

  /* Debugging information about CUDA devices used by the current process */
  if (DebugOutput >= 2)
  {
    printf("Using CUDA Device %s with Properties:\n", deviceProp.name);
    printf("totalGlobalMem = %lld\n",
           (unsigned long long int) deviceProp.totalGlobalMem);
    printf("sharedMemPerBlock = %lld\n",
           (unsigned long long int) deviceProp.sharedMemPerBlock);
    printf("regsPerBlock = %d\n", deviceProp.regsPerBlock);
    printf("warpSize = %d\n", deviceProp.warpSize);
    printf("memPitch = %lld\n", (unsigned long long int) deviceProp.memPitch);
    printf("maxThreadsPerBlock = %d\n", deviceProp.maxThreadsPerBlock);
    printf("maxThreadsDim = %d %d %d\n", deviceProp.maxThreadsDim[0],
           deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
    printf("maxGridSize = %d %d %d\n", deviceProp.maxGridSize[0],
           deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
    printf("totalConstMem = %lld\n",
           (unsigned long long int) deviceProp.totalConstMem);
    printf("major = %d\n", deviceProp.major);
    printf("minor = %d\n", deviceProp.minor);
    printf("clockRate = %d\n", deviceProp.clockRate);
    printf("memoryClockRate = %d\n", deviceProp.memoryClockRate);
    printf("multiProcessorCount = %d\n", deviceProp.multiProcessorCount);
    printf("textureAlignment = %lld\n",
           (unsigned long long int) deviceProp.textureAlignment);
    printf("computeMode = %d\n", deviceProp.computeMode);
#if CUDA_VERSION > 3010
    size_t free, total;
#else
    unsigned int free, total;
#endif
    if (deviceProp.computeMode == 0)
    {
      hipDevice_t tmpDevice;
      cuErrorCheck(hipDeviceGet(&tmpDevice, bestDevice));
      hipCtx_t tmpContext;
      cuErrorCheck(hipCtxCreate(&tmpContext, 0, tmpDevice));
      cuErrorCheck(hipMemGetInfo(&free, &total));
      cuErrorCheck(hipCtxDestroy(tmpContext));
    }
    else
    {
      cuErrorCheck(hipMemGetInfo(&free, &total));
    }
    printf("free memory = %lld; total memory = %lld\n", free, total);
  }

  if (DebugOutput >= 1)
  {
    printf("BioEM for CUDA initialized (MPI Rank %d), %d GPUs found, using GPU "
           "%d\n",
           mpi_rank, count, bestDevice);
  }

  return (0);
}

int bioem_cuda::deviceInit()
{
  deviceExit();

  selectCudaDevice();

  gpumap = new bioem_RefMap;
  memcpy(gpumap, &RefMap, sizeof(bioem_RefMap));

  checkCudaErrors(hipMalloc(&sum, sizeof(myfloat_t) * RefMap.ntotRefMap));
  checkCudaErrors(hipMemcpy(sum, RefMap.sum_RefMap,
                             sizeof(myfloat_t) * RefMap.ntotRefMap,
                             hipMemcpyHostToDevice));
  checkCudaErrors(
      hipMalloc(&sumsquare, sizeof(myfloat_t) * RefMap.ntotRefMap));
  checkCudaErrors(hipMemcpy(sumsquare, RefMap.sumsquare_RefMap,
                             sizeof(myfloat_t) * RefMap.ntotRefMap,
                             hipMemcpyHostToDevice));
  gpumap->sum_RefMap = sum;
  gpumap->sumsquare_RefMap = sumsquare;

  checkCudaErrors(
      hipMalloc(&pProb_memory,
                 pProb_device.get_size(RefMap.ntotRefMap, param.nTotGridAngles,
                                       param.param_device.writeAngles)));

  for (int i = 0; i < PIPELINE_LVL; i++)
  {
    checkCudaErrors(hipStreamCreate(&cudaStream[i]));
    checkCudaErrors(hipEventCreate(&cudaEvent[i]));
  }
  for (int i = 0; i < MULTISTREAM_LVL; i++)
  {
    checkCudaErrors(hipEventCreate(&cudaFFTEvent[i]));
  }
  if (GPUAsync)
  {
    checkCudaErrors(hipStreamCreate(&cudaStream[2]));
    checkCudaErrors(hipEventCreate(&cudaEvent[2]));
  }

  checkCudaErrors(
      hipMalloc(&pRefMapsFFT,
                 RefMap.ntotRefMap * param.FFTMapSize * sizeof(mycomplex_t)));
  checkCudaErrors(
      hipMalloc(&pFFTtmp2[0], param.nTotParallelConv * param.nTotParallelMaps *
                                   param.FFTMapSize * MULTISTREAM_LVL *
                                   sizeof(mycomplex_t)));
  checkCudaErrors(
      hipMalloc(&pFFTtmp[0], param.nTotParallelConv * param.nTotParallelMaps *
                                  param.param_device.NumberPixels *
                                  param.param_device.NumberPixels *
                                  MULTISTREAM_LVL * sizeof(myfloat_t)));
  for (int i = 1; i < MULTISTREAM_LVL; i++)
  {
    pFFTtmp2[i] =
        pFFTtmp2[0] +
        i * param.nTotParallelConv * param.nTotParallelMaps * param.FFTMapSize;
    pFFTtmp[i] = pFFTtmp[0] +
                 i * param.nTotParallelConv * param.nTotParallelMaps *
                     param.param_device.NumberPixels *
                     param.param_device.NumberPixels;
  }
  checkCudaErrors(hipMalloc(&pConvMapFFT, param.nTotParallelConv *
                                               param.FFTMapSize * PIPELINE_LVL *
                                               sizeof(mycomplex_t)));
  checkCudaErrors(hipHostAlloc(&pConvMapFFT_Host,
                                param.nTotParallelConv * param.FFTMapSize *
                                    PIPELINE_LVL * sizeof(mycomplex_t),
                                0));
  checkCudaErrors(
      hipMemcpy(pRefMapsFFT, RefMap.RefMapsFFT,
                 RefMap.ntotRefMap * param.FFTMapSize * sizeof(mycomplex_t),
                 hipMemcpyHostToDevice));
  checkCudaErrors(
      hipMalloc(&pTmp_comp_params,
                 param.nTotParallelConv * PIPELINE_LVL * sizeof(myparam5_t)));
  Ncomp_blocks = divup(param.nTotParallelConv * param.param_device.NtotDisp,
                       CudaThreadCount);
  if (Ncomp_blocks > CudaThreadCount)
  {
    cout << "Error with input parameters. Check CudaThreadCount, "
            "displacements and max number of parallel comparisons\n";
    exit(1);
  }
  checkCudaErrors(
      hipMalloc(&pTmp_comp_blocks,
                 Ncomp_blocks * RefMap.ntotRefMap * sizeof(myblockGPU_t)));

  initialized_const = new bool[RefMap.ntotRefMap];
  for (int i = 0; i < RefMap.ntotRefMap; i++)
    initialized_const[i] = false;

  deviceInitialized = 1;
  return (0);
}

int bioem_cuda::deviceExit()
{
  if (deviceInitialized == 0)
    return (0);

  hipFree(pProb_memory);
  hipFree(sum);
  hipFree(sumsquare);
  for (int i = 0; i < PIPELINE_LVL; i++)
  {
    hipStreamDestroy(cudaStream[i]);
    hipEventDestroy(cudaEvent[i]);
  }
  for (int i = 0; i < MULTISTREAM_LVL; i++)
  {
    hipEventDestroy(cudaFFTEvent[i]);
  }

  hipFree(pRefMapsFFT);
  hipFree(pConvMapFFT);
  hipHostFree(pConvMapFFT_Host);
  hipFree(pFFTtmp[0]);
  hipFree(pFFTtmp2[0]);
  hipFree(pTmp_comp_params);
  hipFree(pTmp_comp_blocks);

  if (GPUAsync)
  {
    hipStreamDestroy(cudaStream[2]);
    hipEventDestroy(cudaEvent[2]);
  }

  delete gpumap;
  delete initialized_const;
  hipDeviceReset();

  deviceInitialized = 0;
  return (0);
}

int bioem_cuda::deviceStartRun()
{
  if (GPUWorkload >= 100)
  {
    maxRef = RefMap.ntotRefMap;
    pProb_host = &pProb;
  }
  else
  {
    maxRef = ((size_t) RefMap.ntotRefMap * (size_t) GPUWorkload / 100) < 1 ?
                 (size_t) RefMap.ntotRefMap :
                 (size_t) RefMap.ntotRefMap * (size_t) GPUWorkload / 100;
    pProb_host = new bioem_Probability;
    pProb_host->init(maxRef, param.nTotGridAngles, *this);
    pProb_host->copyFrom(&pProb, *this);
  }

  pProb_device = *pProb_host;
  pProb_device.ptr = pProb_memory;
  pProb_device.set_pointers();
  checkCudaErrors(
      hipMemcpyAsync(pProb_device.ptr, pProb_host->ptr,
                      pProb_host->get_size(maxRef, param.nTotGridAngles,
                                           param.param_device.writeAngles),
                      hipMemcpyHostToDevice, cudaStream[0]));

  if (maxRef / (param.nTotParallelMaps * param.nTotParallelConv) >
      (double) SPLIT_MAPS_LVL)
  {
    cout << "Error planning CUFFT dimensions\n";
    exit(1);
  }
  for (int j = 0; j < MULTISTREAM_LVL; j++)
  {
    for (int i = 0; i < SPLIT_MAPS_LVL; i++)
    {
      if (i && maxRef % param.nTotParallelMaps == 0)
        continue;
      int n[2] = {param.param_device.NumberPixels,
                  param.param_device.NumberPixels};
      if (hipfftPlanMany(
              &plan[i][j], 2, n, NULL, 1, param.FFTMapSize, NULL, 1, 0,
              MY_CUFFT_C2R,
              i ? ((maxRef % param.nTotParallelMaps) * param.nTotParallelConv) :
                  (param.nTotParallelMaps * param.nTotParallelConv)) !=
          HIPFFT_SUCCESS)
      {
        cout << "Error planning CUFFT\n";
        exit(1);
      }
      if (hipfftSetStream(plan[i][j], cudaStream[j]) != HIPFFT_SUCCESS)
      {
        cout << "Error setting CUFFT stream\n";
        exit(1);
      }
    }
    if (!GPUDualStream)
      break;
  }

  return (0);
}

int bioem_cuda::deviceFinishRun()
{
  if (GPUAsync)
    hipStreamSynchronize(cudaStream[0]);
  checkCudaErrors(
      hipMemcpyAsync(pProb_host->ptr, pProb_device.ptr,
                      pProb_host->get_size(maxRef, param.nTotGridAngles,
                                           param.param_device.writeAngles),
                      hipMemcpyDeviceToHost, cudaStream[0]));

  for (int j = 0; j < MULTISTREAM_LVL; j++)
  {
    for (int i = 0; i < SPLIT_MAPS_LVL; i++)
    {
      if (i && maxRef % param.nTotParallelMaps == 0)
        continue;
      hipfftDestroy(plan[i][j]);
    }
    if (!GPUDualStream)
      break;
  }

  hipDeviceSynchronize();
  if (GPUWorkload < 100)
  {
    pProb.copyFrom(pProb_host, *this);
    free_device_host(pProb_host->ptr);
    delete[] pProb_host;
  }

  return (0);
}

void *bioem_cuda::malloc_device_host(size_t size)
{
  void *ptr;
  checkCudaErrors(hipHostAlloc(&ptr, size, 0));
  return (ptr);
}

void bioem_cuda::free_device_host(void *ptr) { hipHostFree(ptr); }

void bioem_cuda::rebalance(int workload)
{
  if ((workload < 0) || (workload > 100) || (workload == GPUWorkload))
    return;

  deviceFinishRun();

  if (DebugOutput >= 2)
  {
    printf("\t\tSetting GPU workload to %d%% (rank %d)\n", workload, mpi_rank);
  }

  GPUWorkload = workload;
  maxRef = (size_t) RefMap.ntotRefMap * (size_t) GPUWorkload / 100;

  deviceStartRun();
}

bioem *bioem_cuda_create()
{
  int count;

  if (hipGetDeviceCount(&count) != hipSuccess)
    count = 0;
  if (count == 0)
  {
    printf("No CUDA device available, using fallback to CPU version\n");
    return new bioem;
  }

  return new bioem_cuda;
}
